#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 512

struct Drone {
	float x;
	float y;
	float dx;
	float dy;
}

__global__ void flight(float x, float y, float dx, float dy, int n)
{
	int i = threadIdx.x;
	
	x[i] = rand() % 129;
	y[i] = rand() % 129;
	
	//drone.dx = ((rand() % 600 + 30)/3600) * 0.5;	
	
	if(i < n)
	{
		drone.x += dx;
		drone.y += dy;
		if(drone.x >= 128)
		{
			drone.x = drone.x * (-1);
		}		
		if(drone.y >= 128)
		{
			drone.y = drone.y * (-1);
		}
		printf("", drone.x, drone.y);
		
	}
}


int main (void)
{
	//init random numbers
	srand(time(NULL));
	//set up host copies
	float *x, *y, *dx, *dy;	
	//set up device copies
	float *d_x, *d_y, *d_dx, *d_dy;
	int size = sizeof(float);

	//allocate device memory
	hipMalloc((void **), &d_x, size);
	hipMalloc((void **), &d_y, size);
	hipMalloc((void **), &d_dx, size);
	hipMalloc((void **), &d_dy, size);
	
	//allocate host memory
	x = (float *)malloc(size);
	y = (float *)malloc(size);
 	dx = (float *)malloc(size);
	dy = (float *)malloc(size);
	*dx = ((rand() % 601 + 30)/3600) * 0.5;
	*dy = ((rand() % 601 + 30)/3600) * 0.5;

	hipMemcpy(d_dx, dx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dy, dy, size, hipMemcpyHostToDevice);
	
	flight<<<1, N>>>(d_x, d_y, d_dx, d_dy, N);

	free(x); free(y); free(dx); free(dy);

	hipFree(d_x); hipFree(d_y); hipFree(d_dx); hipFree(d_dy);

	return 0;
}
