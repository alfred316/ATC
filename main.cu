#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <unistd.h>

#define N 96
#define pi 3.14159265358979323846

/*
struct Drones
{
float *x;
float *y;
float *dx;
float *dy;
float *nx;
float *ny;

};
*/

//__constant__ float *d_x, *d_y, *d_dx, *d_dy, *d_nx, *d_ny;

//init the positions and velocities of the drones and also calculate their first next position
__global__ void setupFlight(float *x, float *y, float *dx, float *dy, float *nx, float *ny, float *alt, unsigned int seed, hiprandState_t* states)
{
	
	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		threadIdx.x, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[threadIdx.x]);

	int i = threadIdx.x;


	if (i < N)
	{
		//calculate initial pos and velocity
		x[i] = hiprand(&states[threadIdx.x]) % 129;
		y[i] = hiprand(&states[threadIdx.x]) % 129;

		//calculate random speed
		dx[i] = (hiprand(&states[threadIdx.x]) % 601 + 30);
		dy[i] = (hiprand(&states[threadIdx.x]) % 601 + 30);
		//knots per half second
		dx[i] = (dx[i] / 3600) * 0.5;
		dy[i] = (dy[i] / 3600) * 0.5;

		nx[i] = x[i] + dx[i];
		ny[i] = y[i] + dy[i];

		/*
		x[i] += dx[i];
		y[i] += dy[i];
		*/
		
		if (nx[i] >= 128.00)
		{
			nx[i] = nx[i] * (-1.00);
		}
		if (ny[i] >= 128.00)
		{
			ny[i] = ny[i] * (-1.00);
		}

		alt[i] = (hiprand(&states[threadIdx.x]) % 6001 + 3000);
	}
}

//move the aircraft by adding its velocity to it's current location
__global__ void moveDrone(float *x, float *y, float *dx, float *dy, int *col, float *cs, float *sn)
{
	int i = threadIdx.x;
	//create shared variables for next x and y
	//next x
	__shared__ float snx[N];
	//next y
	__shared__ float sny[N];
	//collision
	__shared__ int scol[N];
	//temp x for angle calculation
	__shared__ float spx[N];
	//tmp y for angle calculation
	__shared__ float spy[N];
	//sin * dx for angle calculation on x axis
	__shared__ float tmp[N];
	//cos * dy for angle calculation on y axis
	__shared__ float tmp0[N];

	//upper line x
	__shared__ float upx[N];
	//upper line y
	__shared__ float upy[N];
	//lower like x
	__shared__ float lowx[N];
	//lower line y
	__shared__ float lowy[N];

	//tmp x for batcher's (20 minute path)
	__shared__ float batx[N];
	//tmp y for batcher's
	__shared__ float baty[N];

	//calculate shared values
	if(i < N)
	{
		//calculate next x and y
		snx[i] = x[i] + dx[i];
		sny[i] = y[i] + dy[i];
		//init the shared tmp variables
		spx[i] = 0;
		spy[i] = 0;
		tmp[i] = 0;
		tmp[i] = 0;
		//init the shared upper and lower bounds
		upx[i] = 0;
		upy[i] = 0;
		lowx[i] = 0;
		lowy[i] = 0;
		//init tmp batcher's x and y
		batx[i] = snx[i];
		baty[i] = sny[i];


		//set collision as 0
		scol[i] = 0;
		//if they go out of bounding grid have it come back around
		if (snx[i] >= 128.00)
		{
			snx[i] = snx[i] * (-1.00);
		}
		if (sny[i] >= 128.00)
		{
			sny[i] = sny[i] * (-1.00);
		}

		//wait for calculations to be done on all threads
		__syncthreads();

		//iterate through calculated next x and y's for all drones to see if any collisions happen
		//bounding box is 1 nautical mile on each side (so just +-1 on x and y)

		float bound = 1;

		//iterate through the drones and compare them to the current drone this thread is handling
		//each thread does an interation on all drones, if collision occures, mark both drones as 1
		for (int q = 0; q < N; q++)
		{
			//make sure we are not comparing the same planes with each other
			if (i != q)
			{
				//for loop to check path up to 20 minutes ahead
				//iterate 20 minutes: 20 x 120 half seconds
				//
				int chk = 0;
				for(int t = 0; t < 20 ; t++)
				{
					//every iteration check for collision, if yes, then go back to beginning, fix angle and check again up to 3 times
					if(chk < 3)
					{
						//increase y bounding box each half second by t * dx
						//x + (t * dx) is how far x will be in t half seconds
						//upper bound
						upx[i] = snx[i] + bound + (t * 120 * dx[i]);
						upy[i] = sny[i] + bound + (t * 120 *dy[i]);
						//lower bound
						lowx[i] = snx[i] - bound + (t * 120 * dx[i]);
						lowy[i] = sny[i] - bound + (t * 120 * dy[i]);

						//collision detection and attempt correction based on angle per half second
						if ((batx[q] <= (upx[i])) && (batx[q] >= (lowx[i])))
						{
							//move plane at angle defined in host function
							tmp[i] = *sn * dx[i];
							tmp0[i] = *cs * dy[i];
							spx[i] = batx[i] + tmp[i];
							spy[i] = baty[i] + tmp0[i];
							batx[i] = spx[i];
							baty[i] = spy[i];
							t = -1;
							chk += 1;
						}
						//detection + correction for y bounding box
						if ((baty[q] <= (upy[i])) && (baty[q] >= (lowy[i])))
						{
							//move plane at angle defined in host function
							tmp[i] = *sn * dx[i];
							tmp0[i] = *cs * dy[i];
							spx[i] = batx[i] + tmp[i];
							spy[i] = baty[i] + tmp0[i];
							batx[i] = spx[i];
							baty[i] = spy[i];
							t = -1;
							chk += 1;
						}
					}

				}

				//check if we corrected less than 3 times and got a good path
				//if so, give the batcher's x and y to shared x and y (with their new rotations)
				if(chk < 3)
				{
					snx[i] = batx[i];
					sny[i] = baty[i];
				}

				//old collision detection and correction not using batchers (path ahead)
				//this just checks for immediate bounding box on current position
				/*
				//collision detection and attempt correction based on angle per half second
				if ((snx[q] <= (snx[i] + bound)) && (snx[q] >= (snx[i] - bound)))
				{
					//move plane at angle defined in host function
					tmp[i] = *sn * dx[i];
					tmp0[i] = *cs * dy[i];
					spx[i] = snx[i] + tmp[i];
					spy[i] = sny[i] + tmp0[i];
					snx[i] = spx[i];
					sny[i] = spy[i];
				}
				//detection + correction for y bounding box
				if ((sny[q] <= (sny[i] + bound)) && (sny[q] >= (sny[i] - bound)))
				{
					//move plane at angle defined in host function
					tmp[i] = *sn * dx[i];
					tmp0[i] = *cs * dy[i];
					spx[i] = snx[i] + tmp[i];
					spy[i] = sny[i] + tmp0[i];
					snx[i] = spx[i];
					sny[i] = spy[i];
				}
				*/

				//now after correction attempt check if still colliding
				//check if drone being compared is within our current drone's 1 nm binding box
				if ((snx[q] <= (snx[i] + bound)) && (snx[q] >= (snx[i] - bound)))
				{
					//check that it hasnt been marked 1 before (collided)
					if ((scol[i] == 0) && (scol[q] == 0))
					{
						scol[i] = 1;
						scol[q] = 1;
					}
					
					
				}
				//same as above for y
				else if ((sny[q] <= (sny[i] + bound)) && (sny[q] >= (sny[i] - bound)))
				{
					if ((scol[i] == 0) && (scol[q] == 0))
					{
						scol[i] = 1;
						scol[q] = 1;
					}
					

				}
				else
				{
					scol[i] = 0;
					scol[q] = 0;
				}
			}
		}

		//wait for calculations to be done on all threads
		__syncthreads();

		//give the regular arrays their final values for memcpy by the host
		x[i] = snx[i];
		y[i] = sny[i];
		col[i] = scol[i];

	}

	
}


int main(void)
{
	time_t progStart = time(NULL);
	//Drones *drone_h, *drone_d;
	//init random numbers
	//srand(time(NULL));
	//set up host copies
	float *x, *y, *dx, *dy, *nx, *ny, *alt;
	//set up device copies
	float *d_x, *d_y, *d_dx, *d_dy, *d_nx, *d_ny, *d_alt;
	int size = sizeof(float) * N;
	int *col, *d_col;
	float *angle, *theta, *cs, *sn, *d_cs, *d_sn;

	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
	we will store a random state for every thread  */
	hiprandState_t* states;

	/* allocate space on the GPU for the random states */
	hipMalloc((void**)&states, N * sizeof(hiprandState_t));

	//file
	FILE *file;
	char *fileName = "drones.csv";
	file = fopen(fileName, "w+");
	fprintf(file, "Drone Id, X, Y, dX, dY, Collision\n");

	//allocate host memory
	x = (float*)malloc(size);
	y = (float*)malloc(size);
	dx = (float*)malloc(size);
	dy = (float*)malloc(size);
	nx = (float*)malloc(size);
	ny = (float*)malloc(size);
	alt = (float*)malloc(size);
	col = (int*)malloc(sizeof(int) * N);
	angle = (float*)malloc(size);
	theta = (float*)malloc(size);
	cs = (float*)malloc(size);
	sn = (float*)malloc(size);

	//allocate device memory
	hipMalloc((void**)&d_x, size);
	hipMalloc((void**)&d_y, size);
	hipMalloc((void**)&d_dx, size);
	hipMalloc((void**)&d_dy, size);
	hipMalloc((void**)&d_nx, size);
	hipMalloc((void**)&d_ny, size);
	hipMalloc((void**)&d_alt, size);
	hipMalloc((void**)&d_col, sizeof(int)*N);
	hipMalloc((void**)&d_cs, size);
	hipMalloc((void**)&d_sn, size);

	//hipMalloc((void **)&drone_d, sizeof(Drones)*N);
	//drone_h = (Drones *)malloc(sizeof(Drones)*N);
	
	//angle to rotate planes by every half second
	*angle = 3;
	*theta = (*angle)*((pi) / (180.0));
	*cs = cos(*theta);
	*sn = sin(*theta);

	//*dx = ((rand() % 601 + 30) / 3600) * 0.5;
	//*dy = ((rand() % 601 + 30) / 3600) * 0.5;
	
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dx, dx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_dy, dy, size, hipMemcpyHostToDevice);
	hipMemcpy(d_nx, nx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_ny, ny, size, hipMemcpyHostToDevice);
	hipMemcpy(d_alt, alt, size, hipMemcpyHostToDevice);
	hipMemcpy(d_col, col, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_cs, cs, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sn, sn, size, hipMemcpyHostToDevice);

	//init drones positions, velocity, and first "next position"
	int blocks = 1;
	setupFlight<<<blocks, N >>>(d_x, d_y, d_dx, d_dy, d_nx, d_ny, d_alt, time(NULL), states);


	hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(dx, d_dx, size, hipMemcpyDeviceToHost);
	hipMemcpy(dy, d_dy, size, hipMemcpyDeviceToHost);
	hipMemcpy(nx, d_nx, size, hipMemcpyDeviceToHost);
	hipMemcpy(ny, d_ny, size, hipMemcpyDeviceToHost);
	hipMemcpy(alt, d_alt, size, hipMemcpyDeviceToHost);

	printf("Initializing drones...\n");

	for(int j = 0; j < N; j++)
	{
		printf("Drone #:%d x: %f, y: %f, dx: %f, dy: %f, altitude: %f\n", j, x[j], y[j], dx[j], dy[j], alt[j]);
	}
	
	//radar and collision detection
	//new kernel function to take all these values and keep adding their dx/dy and 
	//checking nx/ny with 1 nautical mile bounding box (+1/-1 on x and y from current x and y)

	printf("Flying drones...\n");

	//each count represents half a second
	float count = 0;
	//duration is how many seconds we want to test this for multiplied by 2
	float duration = 16;

	//loop infinitely until the duration condition is met
	//each iteration run the moveDrone kernel function to move the drone and check collision
	//make sure each iteration takes half a second
	for(;;)
	{
		float interval = 0.5;

		time_t start = time(NULL);
		
		//kernel function and memcpy and printing

		moveDrone << <blocks, N >> >(d_x, d_y, d_dx, d_dy, d_col, d_cs, d_sn);

		hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
		hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
		hipMemcpy(dx, d_dx, size, hipMemcpyDeviceToHost);
		hipMemcpy(dy, d_dy, size, hipMemcpyDeviceToHost);
		hipMemcpy(col, d_col, size, hipMemcpyDeviceToHost);
		hipMemcpy(cs, d_cs, size, hipMemcpyDeviceToHost);
		hipMemcpy(sn, d_sn, size, hipMemcpyDeviceToHost);

		for (int k = 0; k < N; k++)
		{
			printf("Drone #:%d x: %f, y: %f, dx: %f, dy: %f, col: %d\n", k, x[k], y[k], dx[k], dy[k], col[k]);
			fprintf(file, "%d, %f, %f, %f, %f, %d\n", k, x[k], y[k], dx[k], dy[k], col[k]);
		}

		time_t end = time(NULL);
		count += 1;
		if(count >= duration)
		{
			break;
		}

		float elapsed = difftime(end, start);
		float timeLeft = interval - elapsed;
		printf("time left: %f\n", timeLeft);
		if(timeLeft > 0)
		{
			usleep(timeLeft * 1000 * 1000);
		}
	}

	printf("End flight...\n");

	fclose(file);

	time_t progEnd = time(NULL);
	float totalElapsed = difftime(progEnd, progStart);
	printf("total execution time: %f\n", totalElapsed);

	free(x); free(y); free(dx); free(dy); free(nx); free(ny); free(alt); free(col);

	hipFree(d_x); hipFree(d_y); hipFree(d_dx); hipFree(d_dy); hipFree(d_nx); hipFree(d_ny); hipFree(d_alt); hipFree(d_col);

	return 0;
}
