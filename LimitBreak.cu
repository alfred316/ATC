#include "hip/hip_runtime.h"
/*

Alfred Shaker
Air Traffic Control CUDA program
Latest Update: April 1st 2017

*/
/*LIMIT BREAK METHOD*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <windows.h>

#define N 2048 
#define pi 3.14159265358979323846


struct drone
{
float *x; //x coordinate
float *y; //y coordinate
float *batx;//for collision correction batcher's alg
float *baty;//for collision correction batcher's alg
float *dx; //speed on x
float *dy; //speed on y
float *alt; //altitude
int *col; //if on collision course
float *timeTill; //shorted time until next collision
int *colWith; //drone we will collide with
int *rMatchWith; //what radar matched with

} drones[N], sortedDrones[N];

struct radar
{
	float *rx;
	float *ry;
	int *rMatch; //0, 1 or -1 based on not hit, hit, or hit too many times
} radars[N];


//__constant__ float *d_x, *d_y, *d_dx, *d_dy, *d_nx, *d_ny;

//init the positions and velocities of the drones and also calculate their first next position
__global__ void setupFlight(float *x, float *y, float *dx, float *dy, float *alt, int *col, unsigned int seed, hiprandState_t* states)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;


	if (i < N)
	{
		/* we have to initialize the state */
		hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
			i, /* the sequence number is only important with multiple cores */
			0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&states[i]);

		//calculate initial pos and velocity
		x[i] = hiprand(&states[i]) % 129;
		y[i] = hiprand(&states[i]) % 129;

		//calculate random speed
		dx[i] = (hiprand(&states[i]) % 601 + 30);
		dy[i] = (hiprand(&states[i]) % 601 + 30);
		//knots per half second
		dx[i] = (dx[i] / 3600) * 0.5;
		dy[i] = (dy[i] / 3600) * 0.5;

		alt[i] = (hiprand(&states[i]) % 6001 + 3000);

		col[i] = 0;

	}
}

__global__ void GenerateRadarData(float *x, float *y, float *dx, float *dy, float *rx, float *ry, int *rMatch, unsigned int seed, hiprandState_t* states)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		i, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[i]);

	float r = hiprand(&states[i]) / (float)(0x0FFFFFFFFUL);
	float s = hiprand(&states[i]) / (float)(0x0FFFFFFFFUL);


	//create radar by adding random noise to new x and y
	rx[i] = x[i] + dx[i] + r;
	ry[i] = y[i] + dy[i] + s;
	//stores 0, 1 or -1 based on how many planes hit this radar
	rMatch[i] = 0;
}

__global__ void TrackDrone(float *x, float *y, float *dx, float *dy, float *rx, float *ry, int *rMatch, int *rMatchWith)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if(i < N)
	{
	
		float bound = 1;
		int count = 0;

		//move planes 
		x[i] = x[i] + dx[i];
		y[i] = y[i] + dy[i];

		//check boundaries
		if (x[i] >= 128.00)
		{
			x[i] = x[i] * (-1.00);
		}
		if (y[i] >= 128.00)
		{
			y[i] = y[i] * (-1.00);
		}

		//stores the index of the radar corrolated
		rMatchWith[i] = -1;

		__syncthreads();


		//check against all radars to see if we hit any
		for (int p = 0; p < N; p++)
		{
			__syncthreads();
			//check if new x and y are within 1 nautical mile of radar x and y
			if ((x[p] < (rx[i] + bound) && x[p] > (rx[i] - bound)) && (y[p] < (ry[i] + bound) && y[p] > (ry[i] - bound)))
			{
				//once one is found, change the value of srmatch[p] to 1
				//change value of rmatch[i] to p to store the radar we hit and corrolate it to the plane i
				//int set = 0;
				if (rMatch[i] == 0)
				{
					rMatch[i] = 1;
					
				}
				//if a second one is found change srmatch[p] to -1
				else if (rMatch[i] == 1)
				{
					rMatch[i] = -1;
					
				}

				rMatchWith[p] = i;
				__syncthreads();
			}
		}


		//check if there are planes and radars that didnt match, double the bounding box and repeat, up to twice
		//here we go back to having the planes use each thread i index and iterate through the p radars
		if (rMatchWith[i] == -1)
		{
			while (count < 2)
			{
				//check against all radars to see if we hit any
				for (int p = 0; p < N; p++)
				{
					__syncthreads();

					if (rMatch[p] == 0)
					{
						switch (count)
						{
						case 0:
							bound = bound * 2;
							break;

						case 1:
							bound = bound * 4;
							break;
						}
						//check if new x and y are within 1 nautical mile of radar x and y
						if ((x[i] < (rx[p] + bound) && x[i] > (rx[p] - bound)) && (y[i] < (ry[p] + bound) && y[i] > (ry[p] - bound)))
						{
							//once one is found, change the value of srmatch[p] to 1
							//change value of rmatch[i] to p to store the radar we hit and corrolate it to the plane i

							rMatch[p] = 1;

							rMatchWith[i] = p;
							__syncthreads();
						}
					}
				}
				count += 1;
			}
		}

		//check our rMatchWith[i] to see what radar index was hit (where srmatch[i] was not 0)
		//if srmatch[rmatch[i]] == 1, then change new x and y to radar x and y
		//else if srmatch[rmatch[i]] == -1 (hit more than once) or 0 (never hit) then ignore and keep using new x and y
		int m = (int)rMatchWith[i];
		if(m != -1)
		{
			if (rMatch[m] == 1)
			{
				x[i] = rx[m];
				y[i] = ry[m];
				__syncthreads();
			}
		}
		

		//otherwise, use new x and y without radar position

	}
	__syncthreads();
}

//move the aircraft by adding its velocity to it's current location
__global__ void CheckCollisionPath(float *x, float *y, float *batx, float *baty, float *dx, float *dy, int *col, float *timeTill, int * colWith)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	
	float angle, theta, cs, sn;

	//calculate shared values
	if (i < N)
	{
		
		//init timetill and collision with
		timeTill[i] = 300.0;
		colWith[i] = -1;
		col[i] = 0;

		//wait for calculations to be done on all threads
		__syncthreads();

		//iterate through calculated next x and y's for all drones to see if any collisions happen
		//bounding box is 1 nautical mile on each side (so just +-1 on x and y)

		float bound = 1.5; //in nautical miles
		float angleBase = 1; //in degrees


							 //for loop to check path up to 20 minutes ahead
							 //iterate 20 minutes: 20 x 120 half seconds

		int chk;
		chk = 0;
		int t;
		//change to 1 loop starting from 20 ending at more than 20
		for (t = 20; t < 21; t++)
		{
			//every iteration check for collision, if yes, then go back to beginning, fix angle and check again up to 3 times
			if (chk < 3)
			{

				angle = angleBase + (angleBase * chk);
				theta = (angle)*((pi) / (180.0));
				cs = cos(theta);
				sn = sin(theta);

				//get the upper and lower bounds for x and y at the projected time
				//i is our track plane and p is the trial plane


				//may or may not have to correct these paths to go back around if out of bounds. 


				//loop all Q planes and check if any other plane is in our bounding box
				//might be time consuming but will work
				//this is instead of checking against 1 plane 20 times for each device

				//iterate through the drones and compare them to the current drone this thread is handling
				//each thread does an interation on all drones, if collision occures, mark both drones as 1
				for (int p = 0; p < N; ++p)
				{
					//make sure we are not comparing the same planes with each other
					if (i != p)
					{

						//check if any plane already within bounding box on BOTH x and y
						//if ((batx[p] < (batx[i] + bound) && batx[p] > (batx[i] - bound)) && (baty[p] < (baty[i] + bound) && baty[p] > (baty[i] - bound)))
						//{
						//	//move x and y by a certain angle 
						//	batx[i] = cs[i] * x[i] - sn[i] * y[i];
						//	baty[i] = sn[i] * x[i] + cs[i] * y[i];
						//	__syncthreads();

						//	//reset timer and increment check and try again by calling a break out of p loop
						//	t = 19;
						//	chk += 1;
						//	break;
						//}
						//otherwise do batcher's algorithm

						//actual batcher's algorithm code

						float minX, maxX, minY, maxY, timeMin, timeMax;
						float tmpX, tmpY;

						//get min_x
						tmpX = (batx[p] +(t * 120 * dx[i]) )- (batx[i] + (t * 120 * dx[i]));
						minX = fabsf(tmpX);
						minX = minX - 3;
						minX = minX / fabsf(dx[p] - dx[i]);
						//get max_x
						tmpX = (batx[p] + (t * 120 * dx[i])) - (batx[i] + (t * 120 * dx[i]));
						maxX = fabsf(tmpX);
						maxX = maxX + 3;
						maxX = maxX / fabsf(dx[p] - dx[i]);

						//get min_y
						tmpY = (baty[p] + (t * 120 * dy[i])) - (baty[i] + (t * 120 * dy[i]));
						minY = fabsf(tmpY);
						minY = minY - 3;
						minY = minY / fabsf(dy[p] - dy[i]);
						//get max_y
						tmpY = (baty[p] + (t * 120 * dy[i])) - (baty[i] + (t * 120 * dy[i]));
						maxY = fabsf(tmpY);
						maxY = maxY + 3;
						maxY = maxY / fabsf(dy[p] - dy[i]);

						//get time_min and time_max
						timeMin = max(minX, minY);
						timeMax = min(maxX, maxY);

						//check to see if collision will happen on this path
						if (timeMin < timeMax)
						{
							if (timeMin < timeTill[p])
							{
								//update with the soonest collision time 
								timeTill[p] = timeMin;

							}
							if (timeMin < timeTill[i])
							{
								//update with the soonest collision time 
								timeTill[i] = timeMin;

							}

							//set collision variables collision
							col[i] = 1;
							col[p] = 1;
							colWith[i] = p;
							colWith[p] = i;
							__syncthreads();
							//change course and see if we're still on a collision course

							//move x and y by a certain angle 
							batx[i] = cs * x[i] - sn * y[i];
							baty[i] = sn * x[i] + cs * y[i];
							__syncthreads();
							//reset timer and increment check and try again by calling a break out of p loop 
							//and reset t loop back to 20 so repeat the process with new batx and baty and increment check
							t = 19;
							chk += 1;
							break;
						}
						//check if no collision but course corrected AND we have finished checking all planes to assign new x and y based on course correction
						if (chk > 0 && chk < 3 && !(timeMin < timeMax) && p == N - 1)
						{
							x[i] = batx[i];
							y[i] = baty[i];
							col[i] = 0;
							col[p] = 0;
							colWith[i] = -1;
							colWith[p] = -1;
							__syncthreads();
						}
						//else if no collision and no course correction then do nothing to x and y

					}
				}

			}

		}



	}


}


int main(void)
{
	hipEvent_t allStart, allEnd, setupStart, setupEnd, trackingStart, trackingEnd, CollisionDetectionStart, CollisionDetectionEnd;
	hipEventCreate(&allStart);
	hipEventCreate(&allEnd);
	hipEventCreate(&setupStart);
	hipEventCreate(&setupEnd);
	hipEventCreate(&trackingStart);
	hipEventCreate(&trackingEnd);
	hipEventCreate(&CollisionDetectionStart);
	hipEventCreate(&CollisionDetectionEnd);

	hipEventRecord(allStart);

	//time_t progStart = time(NULL);
	//Drones *drone_h, *drone_d;
	//init random numbers
	//srand(time(NULL));
	//set up host copies
	float *x, *y, *batx, *baty, *dx, *dy, *alt, *rx, *ry;
	//set up device copies
	float *d_x, *d_y, *d_dx, *d_dy, *d_alt;
	int size = sizeof(float) * N;
	int *col, *d_col, *rMatch, *rMatchWith;
	float *angle, *theta, *cs, *sn, *d_cs, *d_sn;
	float *h_timeTill, *h_colWith; // *h_upx, *h_upy, *h_lowx, *h_lowy;
	float *d_timeTill, *d_colWith; // *d_upx, *d_upy, *d_lowx, *d_lowy;


								   /* CUDA's random number library uses hiprandState_t to keep track of the seed value
								   we will store a random state for every thread  */
	hiprandState_t* states;

	/* allocate space on the GPU for the random states */
	hipMalloc((void**)&states, N * sizeof(hiprandState_t));

	//file
	FILE *file;
	char *fileName = "drones.csv";
	file = fopen(fileName, "w+");
	fprintf(file, "Drone Id, X, Y, dX, dY, Collision\n");

	//drone *Drones = new drone;

	//allocate host memory
	x = (float*)malloc(size);
	y = (float*)malloc(size);
	dx = (float*)malloc(size);
	dy = (float*)malloc(size);
	alt = (float*)malloc(size);
	col = (int*)malloc(sizeof(int) * N);
	angle = (float*)malloc(size);
	theta = (float*)malloc(size);
	cs = (float*)malloc(size);
	sn = (float*)malloc(size);
	h_timeTill = (float*)malloc(size);
	h_colWith = (float*)malloc(size);
	rx = (float*)malloc(size);
	ry = (float *)malloc(size);
	rMatch = (int *)malloc(sizeof(int) * N);
	rMatchWith = (int *)malloc(sizeof(int) * N);
	batx = (float*)malloc(size);
	baty = (float*)malloc(size);
	/*h_upx = (float*)malloc(size);
	h_upy = (float*)malloc(size);
	h_lowx = (float*)malloc(size);
	h_lowy = (float*)malloc(size);*/

	//allocate device memory
	hipMalloc((void**)&drones->x, size);
	hipMalloc((void**)&drones->y, size);
	hipMalloc((void**)&drones->batx, size);
	hipMalloc((void**)&drones->baty, size);
	hipMalloc((void**)&drones->dx, size);
	hipMalloc((void**)&drones->dy, size);
	hipMalloc((void**)&drones->alt, size);
	hipMalloc((void**)&drones->col, sizeof(int)*N);
	hipMalloc((void**)&d_cs, size);
	hipMalloc((void**)&d_sn, size);
	hipMalloc((void**)&drones->timeTill, size);
	hipMalloc((void**)&drones->colWith, size);
	hipMalloc((void**)&radars->rx, size);
	hipMalloc((void**)&radars->ry, size);
	hipMalloc((void**)&radars->rMatch, sizeof(int)*N);
	hipMalloc((void**)&drones->rMatchWith, sizeof(int)*N);
	/*hipMalloc((void**)&d_upx, size);
	hipMalloc((void**)&d_upy, size);
	hipMalloc((void**)&d_lowx, size);
	hipMalloc((void**)&d_lowy, size);*/

	//hipMalloc((void **)&drone_d, sizeof(Drones)*N);
	//drone_h = (Drones *)malloc(sizeof(Drones)*N);
	
	
	hipMemcpy(drones->x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->y, y, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->batx, batx, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->baty, baty, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->dx, dx, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->dy, dy, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->alt, alt, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->col, col, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_cs, cs, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sn, sn, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->timeTill, h_timeTill, size, hipMemcpyHostToDevice);
	hipMemcpy(drones->colWith, h_colWith, size, hipMemcpyHostToDevice);
	hipMemcpy(radars->rx, rx, size, hipMemcpyHostToDevice);
	hipMemcpy(radars->ry, ry, size, hipMemcpyHostToDevice);
	hipMemcpy(radars->rMatch, rMatch, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(drones->rMatchWith, rMatchWith, sizeof(int)*N, hipMemcpyHostToDevice);
	/*hipMemcpy(d_upx, h_upx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_upy, h_upy, size, hipMemcpyHostToDevice);
	hipMemcpy(d_lowx, h_lowx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_lowy, d_lowy, size, hipMemcpyHostToDevice);*/

	//init drones positions, velocity, and first "next position"
	int blocks = 0;
	int threads = 0;
	if (N < 128)
	{
		blocks = (N + (N - 1)) / N;
		threads = N;
	}
	else
	{
		blocks = (N + 127) / 128;
		threads = 128;
	}

	hipEventRecord(setupStart);

	setupFlight << <blocks, threads >> >(drones->x, drones->y, drones->dx, drones->dy, drones->alt, drones->col, time(NULL), states);

	hipEventRecord(setupEnd);
	hipEventSynchronize(setupEnd);
	float setupTime;
	hipEventElapsedTime(&setupTime, setupStart, setupEnd);
	printf("Setup Flights Time for %d drones: %f ms\n", N, setupTime);


	hipMemcpy(x, drones->x, size, hipMemcpyDeviceToHost);
	hipMemcpy(y, drones->y, size, hipMemcpyDeviceToHost);
	hipMemcpy(dx, drones->dx, size, hipMemcpyDeviceToHost);
	hipMemcpy(dy, drones->dy, size, hipMemcpyDeviceToHost);
	hipMemcpy(alt, drones->alt, size, hipMemcpyDeviceToHost);
	hipMemcpy(col, drones->col, size, hipMemcpyDeviceToHost);

	printf("Initializing drones...\n");

	/*for(int j = 0; j < N; j++)
	{
	printf("Drone #:%d x: %f, y: %f, dx: %f, dy: %f, altitude: %f\n", j, x[j], y[j], dx[j], dy[j], alt[j]);
	}*/

	//radar and collision detection
	//new kernel function to take all these values and keep adding their dx/dy and 
	//checking nx/ny with 1 nautical mile bounding box (+1/-1 on x and y from current x and y)


	/*GENERATE INITIAL RADAR REPORTS*/
	/*RADAR INIT KERNEL*/
	GenerateRadarData << <blocks, threads >> > (drones->x, drones->y, drones->dx, drones->dy, radars->rx, radars->ry, radars->rMatch, time(NULL), states);

	hipMemcpy(rx, radars->rx, size, hipMemcpyDeviceToHost);
	hipMemcpy(ry, radars->ry, size, hipMemcpyDeviceToHost);
	hipMemcpy(rMatch, radars->rMatch, size, hipMemcpyDeviceToHost);

	//reverse each quarter of the array
	//this will simulate a random batch of radars coming in that are not corrolated already by thread id


	int quart = (N / 4);
	float tmpx = 0;
	float tmpy = 0;
	int d = 0;
	int g = 0;
	int v = 0;
	for (v = 0; v < 4; v++)
	{
		d = quart * (1 + v) - 1;
		g = quart * v;
		while (g < d)
		{
			tmpx = rx[g];
			rx[g] = rx[d];
			rx[d] = tmpx;

			tmpy = ry[g];
			ry[g] = ry[d];
			ry[d] = tmpy;

			g++;
			d--;
		}
	}

	printf("Flying drones...\n");

	//each count represents half a second
	int count = 0;
	//duration is how many seconds we want to test this for multiplied by 2
	int duration = 16;

	//loop infinitely until the duration condition is met
	//each iteration run the moveDrone kernel function to move the drone and check collision
	//make sure each iteration takes half a second
	for (;;)
	{
		//interval of half a second, which is 500ms
		float interval = 500.f;

		hipMemcpy(radars->rx, rx, size, hipMemcpyHostToDevice);
		hipMemcpy(radars->ry, ry, size, hipMemcpyHostToDevice);
		hipMemcpy(radars->rMatch, rMatch, sizeof(int)*N, hipMemcpyHostToDevice);
		hipEventRecord(trackingStart);
		//implement tracking with radar in it's own kernel function before collision detection and resolution
		TrackDrone << <blocks, threads >> > (drones->x, drones->y, drones->dx, drones->dy, radars->rx, radars->ry, radars->rMatch, drones->rMatchWith);
		hipEventRecord(trackingEnd);
		hipEventSynchronize(trackingEnd);

		float trackingTime;
		hipEventElapsedTime(&trackingTime, trackingStart, trackingEnd);
		printf("Each Iteration of Flights Tracking Time for %d drones: %f ms\n", N, trackingTime);


		hipMemcpy(x, drones->x, size, hipMemcpyDeviceToHost);
		hipMemcpy(y, drones->y, size, hipMemcpyDeviceToHost);
		hipMemcpy(rx, radars->rx, size, hipMemcpyDeviceToHost);
		hipMemcpy(ry, radars->ry, size, hipMemcpyDeviceToHost);
		hipMemcpy(rMatch, radars->rMatch, sizeof(int)*N, hipMemcpyDeviceToHost);
		hipMemcpy(rMatchWith, drones->rMatchWith, sizeof(int)*N, hipMemcpyDeviceToHost);

		/*for (int k = 0; k < N; k++)
		{
			printf("Drone #:%d x: %f, y: %f, rx: %f, ry: %f, rMatch: %d, rMatchWith: %d\n", k, x[k], y[k], rx[k], ry[k], rMatch[k], rMatchWith[k]);
		}*/

		//time_t start = time(NULL);

		//kernel function and memcpy and printing

		//Collision detection & resolution
		//happens only three times during entire duration

		int half_dur = duration / 2;
		int almostDone = duration - 1;
		float CollisionDetectionTime = 0;

		if (count == 0 || count == half_dur || count == almostDone)
		{
			hipEventRecord(CollisionDetectionStart);
			//add terrain avoidance kernel too somewhere

			//only do collision detection and resolution every few seconds and not every half second step
			CheckCollisionPath << <blocks, threads >> >(drones->x, drones->y, drones->batx, drones->baty, drones->dx, drones->dy, drones->col, drones->timeTill, drones->colWith);

			hipEventRecord(CollisionDetectionEnd);
			hipEventSynchronize(CollisionDetectionEnd);

			hipEventElapsedTime(&CollisionDetectionTime, CollisionDetectionStart, CollisionDetectionEnd);
			printf("Collision detection & resolution for %d drones: %f ms\n", N, CollisionDetectionTime);

			hipMemcpy(x, drones->x, size, hipMemcpyDeviceToHost);
			hipMemcpy(y, drones->y, size, hipMemcpyDeviceToHost);
			hipMemcpy(dx, drones->dx, size, hipMemcpyDeviceToHost);
			hipMemcpy(dy, drones->dy, size, hipMemcpyDeviceToHost);
			hipMemcpy(col, drones->col, size, hipMemcpyDeviceToHost);
			hipMemcpy(h_timeTill, drones->timeTill, size, hipMemcpyDeviceToHost);
			hipMemcpy(h_colWith, drones->colWith, size, hipMemcpyDeviceToHost);

			/*for (int k = 0; k < N; k++)
			{
			printf("Drone #:%d x: %f, y: %f, dx: %f, dy: %f, col: %d\n", k, x[k], y[k], dx[k], dy[k], col[k]);
			fprintf(file, "%d, %f, %f, %f, %f, %d\n", k, x[k], y[k], dx[k], dy[k], col[k]);
			}*/
		}

		/*GENERATE NEW RADAR REPORTS*/
		GenerateRadarData << <blocks, threads >> > (drones->x, drones->y, drones->dx, drones->dy, radars->rx, radars->ry, radars->rMatch, time(NULL), states);

		hipMemcpy(rx, radars->rx, size, hipMemcpyDeviceToHost);
		hipMemcpy(ry, radars->ry, size, hipMemcpyDeviceToHost);

		//reverse each quarter of the array
		//this will simulate a random batch of radars coming in that are not corrolated already by thread id
		//shuffle that shit

		quart = (N / 4);
		tmpx = 0;
		tmpy = 0;
		d = 0;
		g = 0;
		v = 0;
		for (v = 0; v < 4; v++)
		{
			d = quart * (1 + v) - 1;
			g = quart * v;
			while (g < d)
			{
				tmpx = rx[g];
				rx[g] = rx[d];
				rx[d] = tmpx;

				tmpy = ry[g];
				ry[g] = ry[d];
				ry[d] = tmpy;

				g++;
				d--;
			}
		}


		//time_t end = time(NULL);

		count += 1;
		if (count >= duration)
		{
			break;
		}

		//if execution done in under half second, wait the rest of the time to avoid moving
		//more than once every half second
		//elapsed and timeleft are in ms (milliseconds)
		float elapsed = CollisionDetectionTime + trackingTime;
		float timeLeft = interval - elapsed;
		printf("time left: %f\n", timeLeft);
		//makes sure we dont take more than 500ms to execute
		if (timeLeft > 0)
		{
			//sleep works with ms
			Sleep(timeLeft);
		}
	}

	printf("End flight...\n");

	hipEventRecord(allEnd);
	hipEventSynchronize(allEnd);
	float allTime;
	hipEventElapsedTime(&allTime, allStart, allEnd);
	printf("Total Execution Time for %d flights: %f ms\n", N, allTime);

	fclose(file);

	/*
	time_t progEnd = time(NULL);
	float totalElapsed = difftime(progEnd, progStart);
	printf("total execution time: %f\n", totalElapsed);
	*/

	hipEventDestroy(allStart);
	hipEventDestroy(allEnd);
	hipEventDestroy(setupStart);
	hipEventDestroy(setupEnd);
	hipEventDestroy(CollisionDetectionStart);
	hipEventDestroy(CollisionDetectionEnd);

	free(x); free(y); free(dx); free(dy); free(alt); free(col);

	hipFree(drones->x); hipFree(drones->y); hipFree(drones->dx); hipFree(drones->dy); hipFree(drones->alt); hipFree(drones->col);

	return 0;
}
